#include <R.h>
#include <hipfft/hipfft.h>
/* This function is written for R to compute 1D FFT.
   n - [IN] the number of complex we want to compute
   inverse - [IN] set to 1 if use inverse mode
   h_idata_re - [IN] input data from host (R, real part)
   h_idata_im - [IN] input data from host (R, imaginary part)
   h_odata_re - [OUT] results (real) allocated by caller
   h_odata_im - [OUT] results (imaginary) allocated by caller
*/
extern "C"
void cufft(int *n, int *inverse, double *h_idata_re,
           double *h_idata_im, double *h_odata_re, double *h_odata_im)
{
  hipfftHandle plan;
  hipfftDoubleComplex *d_data, *h_data;
  hipMalloc((void**)&d_data, sizeof(hipfftDoubleComplex)*(*n));
  h_data = (hipfftDoubleComplex *) malloc(sizeof(hipfftDoubleComplex) * (*n));

  // Convert data to hipfftDoubleComplex type
  for(int i=0; i< *n; i++) {
    h_data[i].x = h_idata_re[i];
    h_data[i].y = h_idata_im[i];
  }
 
  hipMemcpy(d_data, h_data, sizeof(hipfftDoubleComplex) * (*n), 
             hipMemcpyHostToDevice);
  // Use the CUFFT plan to transform the signal in place.
  hipfftPlan1d(&plan, *n, HIPFFT_Z2Z, 1);
  if (!*inverse ) {
    hipfftExecZ2Z(plan, d_data, d_data, HIPFFT_FORWARD);
  } else {
    hipfftExecZ2Z(plan, d_data, d_data, HIPFFT_BACKWARD);
  }

  hipMemcpy(h_data, d_data, sizeof(hipfftDoubleComplex) * (*n), 
  hipMemcpyDeviceToHost);
  // split hipfftDoubleComplex to double array
  for(int i=0; i<*n; i++) {
    h_odata_re[i] = h_data[i].x;
    h_odata_im[i] = h_data[i].y;
  }
 
  // Destroy the CUFFT plan and free memory.
  hipfftDestroy(plan);
  hipFree(d_data);
  free(h_data);
}